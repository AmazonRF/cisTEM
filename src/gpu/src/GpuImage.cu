#include "hip/hip_runtime.h"
/*
 * GpuImage.cpp
 *
 *  Created on: Jul 31, 2019
 *      Author: himesb
 */

#include "gpu_core_headers.h"


//#include <Exceptions.h>
//#include <helper_string.h>
// Kernel declarations



__global__ void ConvertToHalfPrecisionKernel(hipfftComplex* complex_32f_values, __half2* complex_16f_values, int4 dims, int3 physical_upper_bound_complex);

__global__ void MultiplyPixelWiseComplexConjugateKernel(hipfftComplex* ref_complex_values, hipfftComplex* img_complex_values, int4 dims, int3 physical_upper_bound_complex);
__global__ void MipPixelWiseKernel(hipfftReal *mip, const hipfftReal *correlation_output, const int4 dims);
__global__ void MipPixelWiseKernel(hipfftReal* mip, hipfftReal *other_image, hipfftReal *psi, hipfftReal *phi, hipfftReal *theta,
                                   int4 dims,float c_psi, float c_phi, float c_theta);
__global__ void MipPixelWiseKernel(hipfftReal* mip, hipfftReal *other_image, hipfftReal *psi, hipfftReal *phi, hipfftReal *theta, hipfftReal *defocus, hipfftReal *pixel, const int4 dims,
                                   float c_psi, float c_phi, float c_theta, float c_defocus, float c_pixel);



__global__ void PhaseShiftKernel(hipfftComplex* d_input, 
                                 int4 dims, float3 shifts, 
                                 int3 physical_address_of_box_center, 
                                 int3 physical_index_of_first_negative_frequency,
                                 int3 physical_upper_bound_complex);

__global__ void ClipIntoRealKernel(hipfftReal* real_values_gpu,
                                   hipfftReal* other_image_real_values_gpu,
                                   int4 dims, 
                                   int4 other_dims,
                                   int3 physical_address_of_box_center, 
                                   int3 other_physical_address_of_box_center, 
                                   int3 wanted_coordinate_of_box_center, 
                                   float wanted_padding_value);

// cuFFT callbacks
__device__ hipfftReal CB_ConvertInputf16Tof32(void* dataIn, size_t offset, void* callerInfo, void* sharedPtr);

__device__ hipfftReal CB_ConvertInputf16Tof32(void* dataIn, size_t offset, void* callerInfo, void* sharedPtr)
{

	const __half element = ((__half*)dataIn)[offset];
	return (hipfftReal)(__half2float(element));
}
__device__ hipfftCallbackLoadR d_ConvertInputf16Tof32Ptr = CB_ConvertInputf16Tof32;

__device__ void CB_scaleFFTAndStore(void* dataOut, size_t offset, hipfftComplex element, void* callerInfo, void* sharedPtr);

__device__ void CB_scaleFFTAndStore(void* dataOut, size_t offset, hipfftComplex element, void* callerInfo, void* sharedPtr)
{
	float scale_factor = *(float *)callerInfo;

	((hipfftComplex*)dataOut)[offset] = (hipfftComplex)ComplexScale(element, scale_factor);
}
//__device__ hipfftCallbackLoadR d_ConvertInputf16Tof32Ptr = CB_ConvertInputf16Tof32;
__device__ hipfftCallbackStoreC d_scaleFFTAndStorePtr = CB_scaleFFTAndStore;



//static __device__ hipfftComplex CB_complexConjMulLoad(void* dataIn, size_t offset, void* callerInfo, void* sharedPtr);

template<typename T> struct CB_complexConjMulLoad_params
{
	T* 		target;
	float 	scale;

} ;


static __device__ hipfftComplex CB_complexConjMulLoad_32f(void* dataIn, size_t offset, void* callerInfo, void* sharedPtr);
static __device__ hipfftComplex CB_complexConjMulLoad_16f(void* dataIn, size_t offset, void* callerInfo, void* sharedPtr);

static __device__ hipfftComplex CB_complexConjMulLoad_32f(void* dataIn, size_t offset, void* callerInfo, void* sharedPtr)
 {
	CB_complexConjMulLoad_params<hipfftComplex>* my_params = (CB_complexConjMulLoad_params<hipfftComplex> *)callerInfo;
	return (hipfftComplex)ComplexConjMulAndScale(my_params->target[offset],((Complex *)dataIn)[offset],my_params->scale);
 }
static __device__ hipfftComplex CB_complexConjMulLoad_16f(void* dataIn, size_t offset, void* callerInfo, void* sharedPtr)
 {
	CB_complexConjMulLoad_params<__half2>* my_params = (CB_complexConjMulLoad_params<__half2> *)callerInfo;
	return (hipfftComplex)ComplexConjMulAndScale((hipfftComplex)__half22float2(my_params->target[offset]),((Complex *)dataIn)[offset],my_params->scale);
 }

 __device__ hipfftCallbackLoadC d_complexConjMulLoad_32f = CB_complexConjMulLoad_32f;
 __device__ hipfftCallbackLoadC d_complexConjMulLoad_16f = CB_complexConjMulLoad_16f;

typedef struct _CB_realLoadAndClipInto_params
{
	hipfftReal*	target;
	int i_lower_bound;
	int j_lower_bound;
	int i_upper_bound;
	int j_upper_bound;
	int nx_small;
	int nx_large;

} CB_realLoadAndClipInto_params;


static __device__ hipfftReal CB_realLoadAndClipInto(void* dataIn, size_t offset, void* callerInfo, void* sharedPtr);

static __device__ hipfftReal CB_realLoadAndClipInto(void* dataIn, size_t offset, void* callerInfo, void* sharedPtr)
{
 CB_realLoadAndClipInto_params* my_params = (CB_realLoadAndClipInto_params *)callerInfo;



 int i = offset % my_params->nx_large;
 if (i < my_params->i_lower_bound || i > my_params->i_upper_bound)
 {
	 return (hipfftReal)0.0f;
 }

 int j = (offset-i) / my_params->nx_large;

 if (j < my_params->j_lower_bound || j > my_params->j_upper_bound)
 {
	 return (hipfftReal)0.0f;
 }


 // If we are here, we are in bounds.
 int i_small = i - my_params->i_lower_bound;
 int j_small = j - my_params->j_lower_bound ;
 return my_params->target[i_small + j_small*my_params->nx_small];


}

  __device__ hipfftCallbackLoadR d_realLoadAndClipInto = CB_realLoadAndClipInto;



 // Inline declarations
  __device__ __forceinline__ int
d_ReturnFourierLogicalCoordGivenPhysicalCoord_X(int physical_index, 
                                                int logical_x_dimension,
                                                int physical_address_of_box_center_x);

 __device__ __forceinline__ int
d_ReturnFourierLogicalCoordGivenPhysicalCoord_Y(int physical_index,
                                                int logical_y_dimension,
                                                int physical_index_of_first_negative_frequency_y);

 __device__ __forceinline__ int
d_ReturnFourierLogicalCoordGivenPhysicalCoord_Z(int physical_index,
                                                int logical_z_dimension,
                                                int physical_index_of_first_negative_frequency_x);

 __device__ __forceinline__ float
d_ReturnPhaseFromShift(float real_space_shift, float distance_from_origin, float dimension_size);

 __device__ __forceinline__ void
d_Return3DPhaseFromIndividualDimensions( float phase_x, float phase_y, float phase_z, float2 &angles);

 __device__ __forceinline__ int
d_ReturnReal1DAddressFromPhysicalCoord(int3 coords, int4 img_dims);

 __device__ __forceinline__ int
d_ReturnFourier1DAddressFromPhysicalCoord(int3 wanted_dims, int3 physical_upper_bound_complex);

 __inline__ int
ReturnFourierLogicalCoordGivenPhysicalCoord_Y(int physical_index,
                                              int logical_y_dimension,
                                              int physical_index_of_first_negative_frequency_y)
{
    if (physical_index >= physical_index_of_first_negative_frequency_y)
    {
    	 return physical_index - logical_y_dimension;
    }
    else return physical_index;
};
__inline__ int
ReturnFourierLogicalCoordGivenPhysicalCoord_Z(int physical_index,
                                              int logical_z_dimension,
                                              int physical_index_of_first_negative_frequency_z)
{
    if (physical_index >= physical_index_of_first_negative_frequency_z)
    {
    	 return physical_index - logical_z_dimension;
    }
    else return physical_index;
};

////////////////// For thrust
typedef struct 
{
  __host__ __device__
    float operator()(const float& x) const {
      return x * x;
    }
} square;
////////////////////////

GpuImage::GpuImage()
{ 
  SetupInitialValues();
}


GpuImage::GpuImage(Image &cpu_image) 
{

  SetupInitialValues();
  Init(cpu_image);
	
}

GpuImage::GpuImage( const GpuImage &other_gpu_image) // copy constructor
{

	SetupInitialValues();
	*this = other_gpu_image;
}

GpuImage & GpuImage::operator = (const GpuImage &other_gpu_image)
{
	*this = &other_gpu_image;
	return *this;
}


GpuImage & GpuImage::operator = (const GpuImage *other_gpu_image)
{
	// Check for self assignment
	if(this != other_gpu_image)
	{

		MyAssertTrue(other_gpu_image->is_in_memory_gpu, "Other image Memory not allocated");

		if (is_in_memory_gpu == true)
		{

			if (dims.x != other_gpu_image->dims.x || dims.y != other_gpu_image->dims.y || dims.z != other_gpu_image->dims.z)
			{
				Deallocate();
				Allocate(other_gpu_image->dims.x, other_gpu_image->dims.y, other_gpu_image->dims.z, other_gpu_image->is_in_real_space);
			}
		}
		else
		{
			Allocate(other_gpu_image->dims.x, other_gpu_image->dims.y, other_gpu_image->dims.z, other_gpu_image->is_in_real_space);
		}

		// by here the memory allocation should be ok..

		is_in_real_space = other_gpu_image->is_in_real_space;
		object_is_centred_in_box = other_gpu_image->object_is_centred_in_box;

		pre_checkErrorsAndTimingWithSynchronization(hipStreamPerThread);
		checkCudaErrors(hipMemcpyAsync(real_values_gpu,other_gpu_image->real_values_gpu,sizeof(hipfftReal)*real_memory_allocated,hipMemcpyDeviceToDevice,hipStreamPerThread));
		checkErrorsAndTimingWithSynchronization(hipStreamPerThread);
	}

   return *this;
}

GpuImage::~GpuImage() 
{
//  Deallocate();
}


void GpuImage::Init(Image &cpu_image)
{
	CopyFromCpuImage(cpu_image);
}

void GpuImage::SetupInitialValues()
{

	dims = make_int4(0, 0, 0, 0); pitch = 0;
	physical_upper_bound_complex = make_int3(0, 0, 0);
	physical_address_of_box_center = make_int3(0, 0, 0);
	physical_index_of_first_negative_frequency = make_int3(0, 0, 0);
	logical_upper_bound_complex = make_int3(0, 0, 0);
	logical_lower_bound_complex = make_int3(0, 0, 0);
	logical_upper_bound_real = make_int3(0, 0, 0);
	logical_lower_bound_real = make_int3(0, 0, 0);

	fourier_voxel_size = make_float3(0.0f, 0.0f, 0.0f);


	number_of_real_space_pixels = 0;


	real_values = NULL;
	complex_values = NULL;

	real_memory_allocated = 0;


//	plan_fwd = NULL;
//	plan_bwd = NULL;
//
//	planned = false;

	padding_jump_value = 0;

	ft_normalization_factor = 0;

	real_values_gpu = NULL;									// !<  Real array to hold values for REAL images.
	complex_values_gpu = NULL;								// !<  Complex array to hold values for COMP images.


	gpu_plan_id = -1;

	insert_into_which_reconstruction = 0;
	hostImage = NULL;

	UpdateBoolsToDefault();

}

void GpuImage::CopyFromCpuImage(Image &cpu_image) 
{

	UpdateBoolsToDefault();

	dims = make_int4(cpu_image.logical_x_dimension,
				   cpu_image.logical_y_dimension,
				   cpu_image.logical_z_dimension,
				   cpu_image.logical_x_dimension + cpu_image.padding_jump_value);

	pitch = dims.w * sizeof(float);

	physical_upper_bound_complex = make_int3(cpu_image.physical_upper_bound_complex_x,
										   cpu_image.physical_upper_bound_complex_y,
										   cpu_image.physical_upper_bound_complex_z);

	physical_address_of_box_center   = make_int3(cpu_image.physical_address_of_box_center_x,
											   cpu_image.physical_address_of_box_center_y,
											   cpu_image.physical_address_of_box_center_z);

	physical_index_of_first_negative_frequency = make_int3(0,
														 cpu_image.physical_index_of_first_negative_frequency_y,
														 cpu_image.physical_index_of_first_negative_frequency_z);


	logical_upper_bound_complex = make_int3(cpu_image.logical_upper_bound_complex_x,
										  cpu_image.logical_upper_bound_complex_y,
										  cpu_image.logical_upper_bound_complex_z);


	logical_lower_bound_complex = make_int3(cpu_image.logical_lower_bound_complex_x,
										  cpu_image.logical_lower_bound_complex_y,
										  cpu_image.logical_lower_bound_complex_z);


	logical_upper_bound_real = make_int3(cpu_image.logical_upper_bound_real_x,
									   cpu_image.logical_upper_bound_real_y,
									   cpu_image.logical_upper_bound_real_z);

	logical_lower_bound_real = make_int3(cpu_image.logical_lower_bound_real_x,
									   cpu_image.logical_lower_bound_real_y,
									   cpu_image.logical_lower_bound_real_z);


	is_in_real_space = cpu_image.is_in_real_space;
	number_of_real_space_pixels = cpu_image.number_of_real_space_pixels;
	object_is_centred_in_box = cpu_image.object_is_centred_in_box;

	fourier_voxel_size = make_float3(cpu_image.fourier_voxel_size_x,
								   cpu_image.fourier_voxel_size_y,
								   cpu_image.fourier_voxel_size_z);


	insert_into_which_reconstruction = cpu_image.insert_into_which_reconstruction;
	real_values = cpu_image.real_values;
	complex_values = cpu_image.complex_values;

	is_in_memory = cpu_image.is_in_memory;

	padding_jump_value = cpu_image.padding_jump_value;
	image_memory_should_not_be_deallocated = cpu_image.image_memory_should_not_be_deallocated; // TODO what is this for?

	real_values_gpu = NULL;									// !<  Real array to hold values for REAL images.
	complex_values_gpu = NULL;								// !<  Complex array to hold values for COMP images.
	is_in_memory_gpu = false;
	real_memory_allocated =  cpu_image.real_memory_allocated;


	ft_normalization_factor = cpu_image.ft_normalization_factor;

	// FIXME for now always pin the memory - this might be a bad choice for single copy or small images, but is required for asynch xfer and is ~2x as fast after pinning
	hipHostRegister(real_values, sizeof(float)*real_memory_allocated, hipHostRegisterDefault);
	is_host_memory_pinned = true;
	is_meta_data_initialized = true;
	hipHostGetDevicePointer( &pinnedPtr, real_values, 0);

	hipMallocManaged(&tmpVal, sizeof(hipfftReal));
	hipMallocManaged(&tmpValComplex, sizeof(hipfftComplex));

	hostImage = &cpu_image;
 
}

void GpuImage::UpdateCpuFlags() 
{

  // Call after re-copying. The main image properites are all assumed to be static.
  is_in_real_space = hostImage->is_in_real_space;
  object_is_centred_in_box = hostImage->object_is_centred_in_box;

}

void GpuImage::printVal(std::string msg, int idx)
{

  float h_printVal = -9999.0f;

  checkCudaErrors(hipMemcpy(&h_printVal, &real_values_gpu[idx], sizeof(float), hipMemcpyDeviceToHost));
  hipStreamSynchronize(hipStreamPerThread);
  wxPrintf("%s %6.6e\n", msg, h_printVal);

};

bool GpuImage::HasSameDimensionsAs(GpuImage *other_image)
{
	// Functions that call this method also assume these asserts are being called here, so do not remove.
	MyAssertTrue(is_in_memory_gpu, "Memory not allocated");
	MyAssertTrue(other_image->is_in_memory_gpu, "Other image Memory not allocated");
	// end of dependent asserts.

	if (dims.x == other_image->dims.x && dims.y == other_image->dims.y && dims.z == other_image->dims.z) return true;
	else return false;
}

void GpuImage::MultiplyPixelWiseComplexConjugate(GpuImage &other_image)
{
	// FIXME when adding real space complex images
	MyAssertFalse( is_in_real_space, "Image is in real space");
	MyAssertFalse( other_image.is_in_real_space, "Other image is in real space");
	MyAssertTrue(HasSameDimensionsAs(&other_image), "Images have different dimensions");


	//  NppInit();
	//  Conj();
	//  npp_stat = nppiMul_32sc_C1IRSfs((const Npp32sc *)complex_values_gpu, 1, (Npp32sc*)other_image.complex_values_gpu, 1, npp_ROI_complex, 0);

	pre_checkErrorsAndTimingWithSynchronization(hipStreamPerThread);
	ReturnLaunchParamters(dims, false);
	MultiplyPixelWiseComplexConjugateKernel<< <gridDims, threadsPerBlock,0, hipStreamPerThread>> > (complex_values_gpu, other_image.complex_values_gpu,this->dims, this->physical_upper_bound_complex);
	checkErrorsAndTimingWithSynchronization(hipStreamPerThread);

}


__global__ void ReturnSumOfRealValuesOnEdgesKernel(hipfftReal *real_values_gpu, int4 dims, int padding_jump_value, float* returnValue);

float GpuImage::ReturnAverageOfRealValuesOnEdges()
{
	// FIXME to use a masked routing, this is slow af
	MyAssertTrue(is_in_memory, "Memory not allocated");
	MyAssertTrue(dims.z == 1, "ReturnAverageOfRealValuesOnEdges only implemented in 2d");

	pre_checkErrorsAndTimingWithSynchronization(hipStreamPerThread);
	*tmpVal = 5.0f;
	ReturnSumOfRealValuesOnEdgesKernel<< <1, 1, 0, hipStreamPerThread>> >(real_values_gpu, dims, padding_jump_value, tmpVal);
	checkErrorsAndTimingWithSynchronization(hipStreamPerThread);

	// Need to wait on the return value
	checkCudaErrors(hipStreamSynchronize(hipStreamPerThread));


  return *tmpVal;
}

__global__ void ReturnSumOfRealValuesOnEdgesKernel(hipfftReal *real_values_gpu, int4 dims, int padding_jump_value, float* returnValue)
{

	int pixel_counter;
	int line_counter;
	int plane_counter;

	double sum = 0.0;
	int number_of_pixels = 0;
	int address = 0;


		// Two-dimensional image
		// First line
		for (pixel_counter=0; pixel_counter < dims.x; pixel_counter++)
		{
			sum += real_values_gpu[address];
			address++;
		}
		number_of_pixels += dims.x;
		address += padding_jump_value;

		// Other lines
		for (line_counter=1; line_counter < dims.y-1; line_counter++)
		{
			sum += real_values_gpu[address];
			address += dims.x-1;
			sum += real_values_gpu[address];
			address += padding_jump_value + 1;
			number_of_pixels += 2;
		}

		// Last line
		for (pixel_counter=0; pixel_counter < dims.x; pixel_counter++)
		{
			sum += real_values_gpu[address];
			address++;
		}
		number_of_pixels += dims.x;

   *returnValue = (float)sum / (float)number_of_pixels;
}

//void GpuImage::CublasInit()
//{
//  if ( ! is_cublas_loaded )
//  {
//    hipblasCreate(&cublasHandle);
//    is_cublas_loaded = true;
//    hipblasSetStream(cublasHandle, hipStreamPerThread);
//  }
//}

void GpuImage::NppInit()
{
  if ( ! is_npp_loaded )
  {

	int sharedMem;
	nppStream.hStream = hipStreamPerThread; // FIXME to use member stream
	hipGetDevice(&nppStream.nCudaDeviceId);
	hipDeviceGetAttribute(&nppStream.nMultiProcessorCount,hipDeviceAttributeMultiprocessorCount,nppStream.nCudaDeviceId);
	hipDeviceGetAttribute(&nppStream.nMaxThreadsPerMultiProcessor,hipDeviceAttributeMaxThreadsPerMultiProcessor,nppStream.nCudaDeviceId);
	hipDeviceGetAttribute(&nppStream.nMaxThreadsPerBlock,hipDeviceAttributeMaxThreadsPerBlock,nppStream.nCudaDeviceId);
	hipDeviceGetAttribute(&nppStream.nMaxThreadsPerMultiProcessor,hipDeviceAttributeMaxThreadsPerMultiProcessor,nppStream.nCudaDeviceId);
	hipDeviceGetAttribute(&sharedMem,hipDeviceAttributeMaxSharedMemoryPerBlock,nppStream.nCudaDeviceId);
	nppStream.nSharedMemPerBlock = (size_t)sharedMem;
	hipDeviceGetAttribute(&nppStream.nCudaDevAttrComputeCapabilityMajor,hipDeviceAttributeComputeCapabilityMajor,nppStream.nCudaDeviceId);
	hipDeviceGetAttribute(&nppStream.nCudaDevAttrComputeCapabilityMinor,hipDeviceAttributeComputeCapabilityMinor,nppStream.nCudaDeviceId);

//    nppSetStream(hipStreamPerThread);
    npp_ROI.width  = dims.x;
    npp_ROI.height = dims.y * dims.z;

    npp_ROI_complex.width = dims.w / 2;
    npp_ROI_complex.height = dims.y * dims.z;

    is_npp_loaded = true;

  }
}

void GpuImage::BufferInit(BufferType bt)
{

  switch (bt)
  {
    case b_image :
        if ( ! is_allocated_image_buffer )
        {
          image_buffer = new GpuImage;
          *image_buffer = *this;
          is_allocated_image_buffer = true;
        }     
        break;

    case b_16f :
    	if ( ! is_allocated_16f_buffer )
    	{
    		checkCudaErrors(hipMalloc(&real_values_16f, sizeof(__half)*real_memory_allocated));
    		complex_values_16f = (__half2 *)real_values_16f;
    		is_allocated_16f_buffer = true;
    	}
    	break;

    case b_sum :
        if ( ! is_allocated_sum_buffer ) 
        {
          int n_elem;
          nppiSumGetBufferHostSize_32f_C1R_Ctx(npp_ROI, &n_elem,nppStream);
          checkCudaErrors(hipMalloc(&this->sum_buffer, n_elem));
          is_allocated_sum_buffer = true;
        }     
        break;   

    case b_min :
        if ( ! is_allocated_min_buffer )
        {
          int n_elem;
          nppiMinGetBufferHostSize_32f_C1R_Ctx(npp_ROI, &n_elem,nppStream);
          checkCudaErrors(hipMalloc(&this->min_buffer, n_elem));
          is_allocated_min_buffer = true;
        }
        break;

  case b_minIDX :
      if ( ! is_allocated_minIDX_buffer )
      {
        int n_elem;
        nppiMinIndxGetBufferHostSize_32f_C1R_Ctx(npp_ROI, &n_elem,nppStream);
        checkCudaErrors(hipMalloc(&this->minIDX_buffer, n_elem));
        is_allocated_minIDX_buffer = true;
      }
      break;

  case b_max :
      if ( ! is_allocated_max_buffer )
      {
        int n_elem;
        nppiMaxGetBufferHostSize_32f_C1R_Ctx(npp_ROI, &n_elem,nppStream);
        checkCudaErrors(hipMalloc(&this->max_buffer, n_elem));
        is_allocated_max_buffer = true;
      }
      break;

  case b_maxIDX :
      if ( ! is_allocated_maxIDX_buffer )
      {
        int n_elem;
        nppiMaxIndxGetBufferHostSize_32f_C1R_Ctx(npp_ROI, &n_elem,nppStream);
        checkCudaErrors(hipMalloc(&this->maxIDX_buffer, n_elem));
        is_allocated_maxIDX_buffer = true;
      }
      break;

  case b_minmax :
      if ( ! is_allocated_minmax_buffer )
      {
        int n_elem;
        nppiMinMaxGetBufferHostSize_32f_C1R_Ctx(npp_ROI, &n_elem,nppStream);
        checkCudaErrors(hipMalloc(&this->minmax_buffer, n_elem));
        is_allocated_minmax_buffer = true;
      }
      break;

  case b_minmaxIDX :
      if ( ! is_allocated_minmaxIDX_buffer )
      {
        int n_elem;
        nppiMinMaxIndxGetBufferHostSize_32f_C1R_Ctx(npp_ROI, &n_elem,nppStream);
        checkCudaErrors(hipMalloc(&this->minmaxIDX_buffer, n_elem));
        is_allocated_minmaxIDX_buffer = true;
      }
      break;

  case b_mean :
      if ( ! is_allocated_mean_buffer )
      {
        int n_elem;
        nppiMeanGetBufferHostSize_32f_C1R_Ctx(npp_ROI, &n_elem,nppStream);
        checkCudaErrors(hipMalloc(&this->mean_buffer, n_elem));
        is_allocated_mean_buffer = true;
      }
      break;

  case b_meanstddev :
      if ( ! is_allocated_meanstddev_buffer )
      {
        int n_elem;
        nppiMeanGetBufferHostSize_32f_C1R_Ctx(npp_ROI, &n_elem,nppStream);
        checkCudaErrors(hipMalloc(&this->meanstddev_buffer, n_elem));
        is_allocated_meanstddev_buffer = true;
      }
      break;

  case b_countinrange :
      if ( ! is_allocated_countinrange_buffer )
      {
        int n_elem;
        nppiCountInRangeGetBufferHostSize_32f_C1R_Ctx(npp_ROI, &n_elem,nppStream);
        checkCudaErrors(hipMalloc(&this->countinrange_buffer, n_elem));
        is_allocated_countinrange_buffer = true;
      }
      break;

  case b_l2norm :
	  if (! is_allocated_l2norm_buffer)
	  {
		  int n_elem;
		  nppiNormL2GetBufferHostSize_32f_C1R_Ctx(npp_ROI, &n_elem,nppStream);
		  checkCudaErrors(hipMalloc(&this->l2norm_buffer,n_elem));

		  is_allocated_l2norm_buffer = true;
	  }
	  break;

  case b_dotproduct :
	  if ( ! is_allocated_dotproduct_buffer )
	  {
		  int n_elem;
		  nppiDotProdGetBufferHostSize_32f64f_C1R_Ctx(npp_ROI, &n_elem, nppStream);
		  checkCudaErrors(hipMalloc(&this->dotproduct_buffer, n_elem));
		  is_allocated_dotproduct_buffer = true;
	  }
	  break;


}

}


float GpuImage::ReturnSumOfSquares()
{

	// FIXME this assumes padded values are zero which is not strictly true
	MyAssertTrue(is_in_memory_gpu, "Image not allocated");
	MyAssertTrue(is_in_real_space, "This method is for real space, use ReturnSumSquareModulusComplexValues for Fourier space")


//	float returnValue = 0.0f;
	Npp64f* pNorm;
	checkCudaErrors(hipMallocManaged(&pNorm, sizeof(Npp64f)));

	BufferInit(b_l2norm);
	NppInit();

	checkNppErrors(nppiNorm_L2_32f_C1R_Ctx((Npp32f *)real_values_gpu, pitch, npp_ROI,
									 	   (Npp64f *)pNorm, (Npp8u *)this->l2norm_buffer, nppStream));

	return ((float)*pNorm * (float)*pNorm);

//	CublasInit();complex_6EK0_SSU.pdb
//	// With real and complex interleaved, treating as real is equivalent to taking the conj dot prod
//	cublas_stat = hipblasSdot( cublasHandle, real_memory_allocated,
//							real_values_gpu, 1,
//							real_values_gpu, 1,
//							&returnValue);
//
//	if (cublas_stat) {
//	wxPrintf("Cublas return val %s\n", cublas_stat); }


//	return returnValue;

}



float GpuImage::ReturnSumSquareModulusComplexValues()
{

	//
	MyAssertTrue(is_in_memory_gpu, "Image not allocated");
	MyAssertFalse(is_in_real_space, "This method is NOT for real space, use ReturnSumofSquares for realspace")
	int address = 0;
	bool x_is_even = IsEven(dims.x);
	int i,j,k,jj,kk;
	const std::complex<float> c1(sqrtf(0.25f),sqrtf(0.25));
	const std::complex<float> c2(sqrtf(0.5),sqrtf(0.5)); // original code is pow(abs(Val),2)*0.5
	const std::complex<float> c3(1.0,1.0);
	const std::complex<float> c4(0.0,0.0);
	float returnValue;

	if ( ! is_allocated_mask_CSOS )
	{

		wxPrintf("is mask allocated %d\n", is_allocated_mask_CSOS);
		mask_CSOS = new GpuImage;
		is_allocated_mask_CSOS = true;
		wxPrintf("is mask allocated %d\n", is_allocated_mask_CSOS);
		// create a mask that can be reproduce the correct weighting from Image::ReturnSumOfSquares on complex images

		wxPrintf("\n\tMaking mask_CSOS\n");
		mask_CSOS->Allocate(dims.x, dims.y, dims.z, true);
		// The mask should always be in real_space, and starts out not centered
		mask_CSOS->is_in_real_space = true;
		mask_CSOS->object_is_centred_in_box = true;
		// Allocate pinned host memb
		checkCudaErrors(hipHostAlloc(&mask_CSOS->real_values, sizeof(float)*real_memory_allocated, hipHostMallocDefault));
		mask_CSOS->complex_values = (std::complex<float>*) mask_CSOS->real_values;
		for (k = 0; k <= physical_upper_bound_complex.z; k++)
		{

			kk = ReturnFourierLogicalCoordGivenPhysicalCoord_Z(k, dims.z, physical_index_of_first_negative_frequency.z);
			for (j = 0; j <= physical_upper_bound_complex.y; j++)
			{
				jj = ReturnFourierLogicalCoordGivenPhysicalCoord_Y(j,dims.y, physical_index_of_first_negative_frequency.y);
				for (i = 0; i <= physical_upper_bound_complex.x; i++)
				{
					if ((i == 0  || (i  == logical_upper_bound_complex.x && x_is_even)) && 
			  (jj == 0 || (jj == logical_lower_bound_complex.y && x_is_even)) &&
			  (kk == 0 || (kk == logical_lower_bound_complex.z && x_is_even)))
		  {
			mask_CSOS->complex_values[address] = c2;

		  }
					else if ((i == 0 || (i == logical_upper_bound_complex.x && x_is_even)) && dims.z != 1) 
		  {
			mask_CSOS->complex_values[address] = c1;
		  }
					else if ((i != 0 && (i != logical_upper_bound_complex.x || ! x_is_even)) || (jj >= 0 && kk >= 0)) 
		  {
			mask_CSOS->complex_values[address] = c3;
		  }
		  else
		  {
			mask_CSOS->complex_values[address] = c4;
		  }

					address++;
				}
			}
		}   

		pre_checkErrorsAndTimingWithSynchronization(hipStreamPerThread);
		checkCudaErrors(hipMemcpyAsync(mask_CSOS->real_values_gpu, mask_CSOS->real_values,sizeof(float)*real_memory_allocated,hipMemcpyHostToDevice,hipStreamPerThread));
		pre_checkErrorsAndTimingWithSynchronization(hipStreamPerThread);
		// TODO change this to an event that can then be later checked prior to deleteing
		checkCudaErrors(hipStreamSynchronize(hipStreamPerThread));
		checkCudaErrors(hipHostFree(mask_CSOS->real_values));

	} // end of mask creation


	BufferInit(b_image);
	pre_checkErrorsAndTimingWithSynchronization(hipStreamPerThread);
	checkCudaErrors(hipMemcpyAsync(image_buffer->real_values_gpu, mask_CSOS->real_values_gpu, sizeof(float)*real_memory_allocated,hipMemcpyDeviceToDevice,hipStreamPerThread));
	checkErrorsAndTimingWithSynchronization(hipStreamPerThread);
	image_buffer->MultiplyPixelWise(*this);

//	CublasInit();
	// With real and complex interleaved, treating as real is equivalent to taking the conj dot prod
	pre_checkErrorsAndTimingWithSynchronization(hipStreamPerThread);

	NppInit();
	BufferInit(b_dotproduct);
	Npp64f* dotProd;
	checkCudaErrors(hipMallocManaged(&dotProd, sizeof(Npp64f)));

	checkNppErrors(nppiDotProd_32f64f_C1R_Ctx((Npp32f *)image_buffer->real_values_gpu, pitch,
											  (Npp32f *)image_buffer->real_values_gpu, pitch,
											  npp_ROI, dotProd, dotproduct_buffer, nppStream));

//	hipblasSdot( cublasHandle, real_memory_allocated,
//			  image_buffer->real_values_gpu, 1,
//			  image_buffer->real_values_gpu, 1,
//			  &returnValue);
	checkErrorsAndTimingWithSynchronization(hipStreamPerThread);

	return (float)(*dotProd * 2.0f);

  
}


__global__ void MultiplyPixelWiseComplexConjugateKernel(hipfftComplex* ref_complex_values, hipfftComplex* img_complex_values, int4 dims, int3 physical_upper_bound_complex)
{
  int3 coords = make_int3(blockIdx.x*blockDim.x + threadIdx.x,
                          blockIdx.y*blockDim.y + threadIdx.y,
                          blockIdx.z);

    if (coords.x < dims.w / 2 && coords.y < dims.y && coords.z < dims.z)
    {

	    int address = d_ReturnFourier1DAddressFromPhysicalCoord(coords, physical_upper_bound_complex);

	    ref_complex_values[address] = (hipfftComplex)ComplexConjMul((Complex)img_complex_values[address],(Complex)ref_complex_values[address]);
    }
    

}

void GpuImage::MipPixelWise(GpuImage &other_image)
{

	MyAssertTrue(HasSameDimensionsAs(&other_image), "Images have different dimension.");
	pre_checkErrorsAndTimingWithSynchronization(hipStreamPerThread);
	ReturnLaunchParamters(dims, true);
	MipPixelWiseKernel<< <gridDims, threadsPerBlock,0,hipStreamPerThread>> > (real_values_gpu, other_image.real_values_gpu, this->dims);
	checkErrorsAndTimingWithSynchronization(hipStreamPerThread);

}
__global__ void MipPixelWiseKernel(hipfftReal *mip, const hipfftReal *correlation_output, const int4 dims)
{

    int3 coords = make_int3(blockIdx.x*blockDim.x + threadIdx.x,
                          blockIdx.y*blockDim.y + threadIdx.y,
                          blockIdx.z);

    if (coords.x < dims.x && coords.y < dims.y && coords.z < dims.z)
    {
	    int address = d_ReturnReal1DAddressFromPhysicalCoord(coords, dims);
	    mip[address] = MAX(mip[address], correlation_output[address]);
    }
}

void GpuImage::MipPixelWise(GpuImage &other_image, GpuImage &psi, GpuImage &phi, GpuImage &theta, GpuImage &defocus, GpuImage &pixel,
                            float c_psi, float c_phi, float c_theta, float c_defocus, float c_pixel)
{

	MyAssertTrue(HasSameDimensionsAs(&other_image), "Images have different dimension.");
	pre_checkErrorsAndTimingWithSynchronization(hipStreamPerThread);
	ReturnLaunchParamters(dims, true);
	MipPixelWiseKernel<< <gridDims, threadsPerBlock,0,hipStreamPerThread>> >(real_values_gpu, other_image.real_values_gpu,
																	   psi.real_values_gpu,phi.real_values_gpu,theta.real_values_gpu,defocus.real_values_gpu,pixel.real_values_gpu,
																		this->dims, c_psi, c_phi, c_theta, c_defocus, c_pixel);
	checkErrorsAndTimingWithSynchronization(hipStreamPerThread);


}



__global__ void MipPixelWiseKernel(hipfftReal* mip, hipfftReal *correlation_output, hipfftReal *psi, hipfftReal *phi, hipfftReal *theta, hipfftReal *defocus, hipfftReal *pixel, const int4 dims,
                                   float c_psi, float c_phi, float c_theta, float c_defocus, float c_pixel)
{

    int3 coords = make_int3(blockIdx.x*blockDim.x + threadIdx.x,
                          blockIdx.y*blockDim.y + threadIdx.y,
                          blockIdx.z);

    if (coords.x < dims.x && coords.y < dims.y && coords.z < dims.z)
    {
	    int address = d_ReturnReal1DAddressFromPhysicalCoord(coords, dims);
      if (correlation_output[address] > mip[address])
      {
        mip[address] = correlation_output[address];
        psi[address] = c_psi;
        phi[address] = c_phi;
        theta[address] = c_theta;
        defocus[address] = c_defocus;
        pixel[address] = c_pixel;
      }

    }
}

void GpuImage::MipPixelWise(GpuImage &other_image, GpuImage &psi, GpuImage &phi, GpuImage &theta,
                            float c_psi, float c_phi, float c_theta)
{

	MyAssertTrue(HasSameDimensionsAs(&other_image), "Images have different dimension.");
	pre_checkErrorsAndTimingWithSynchronization(hipStreamPerThread);
	ReturnLaunchParamters(dims, true);
	MipPixelWiseKernel<< <gridDims, threadsPerBlock,0,hipStreamPerThread>> >(real_values_gpu, other_image.real_values_gpu,
																	   psi.real_values_gpu,phi.real_values_gpu,theta.real_values_gpu,
																	   this->dims, c_psi, c_phi, c_theta);
	checkErrorsAndTimingWithSynchronization(hipStreamPerThread);

}



__global__ void MipPixelWiseKernel(hipfftReal* mip, hipfftReal *correlation_output, hipfftReal *psi, hipfftReal *phi, hipfftReal *theta, const int4 dims,
                                   float c_psi, float c_phi, float c_theta)
{

    int3 coords = make_int3(blockIdx.x*blockDim.x + threadIdx.x,
                          blockIdx.y*blockDim.y + threadIdx.y,
                          blockIdx.z);

    if (coords.x < dims.x && coords.y < dims.y && coords.z < dims.z)
    {
	  int address = d_ReturnReal1DAddressFromPhysicalCoord(coords, dims);
      if (correlation_output[address] > mip[address])
      {
        mip[address] = correlation_output[address];
        psi[address] = c_psi;
        phi[address] = c_phi;
        theta[address] = c_theta;
      }

    }
}


void GpuImage::Abs()
{
	MyAssertTrue(is_in_memory_gpu, "Memory not allocated");

  NppInit();
  checkNppErrors(nppiAbs_32f_C1IR_Ctx((Npp32f *)real_values_gpu, pitch, npp_ROI,nppStream));
}

void GpuImage::AbsDiff(GpuImage &other_image)
{
	MyAssertTrue(HasSameDimensionsAs(&other_image), "Images have different dimension.");


	BufferInit(b_image);
	NppInit();

	checkNppErrors(nppiAbsDiff_32f_C1R_Ctx((const Npp32f *)real_values_gpu, pitch,
									 (const Npp32f *)other_image.real_values_gpu, pitch,
									 (      Npp32f *)this->image_buffer->real_values_gpu, pitch, npp_ROI,nppStream));

	pre_checkErrorsAndTimingWithSynchronization(hipStreamPerThread);
	checkCudaErrors(hipMemcpyAsync(real_values_gpu,this->image_buffer->real_values_gpu,sizeof(hipfftReal)*real_memory_allocated,hipMemcpyDeviceToDevice,hipStreamPerThread));
	checkErrorsAndTimingWithSynchronization(hipStreamPerThread);

}

void GpuImage::AbsDiff(GpuImage &other_image, GpuImage &output_image)
{
  // In place abs diff (see overload for out of place)
	MyAssertTrue(HasSameDimensionsAs(&other_image), "Images have different dimension.");
	MyAssertTrue(HasSameDimensionsAs(&output_image), "Images have different dimension.");



  NppInit();
  
  checkNppErrors(nppiAbsDiff_32f_C1R_Ctx((const Npp32f *)real_values_gpu, pitch,
                                     (const Npp32f *)other_image.real_values_gpu, pitch,
                                     (      Npp32f *)output_image.real_values_gpu, pitch, npp_ROI,nppStream));

}

void GpuImage::Min()
{
	MyAssertTrue(is_in_memory_gpu, "Memory not allocated");
	MyAssertTrue(is_in_real_space, "Not in real space");

	NppInit();
	BufferInit(b_min);
	checkNppErrors(nppiMin_32f_C1R_Ctx((const Npp32f*)real_values_gpu, pitch, npp_ROI, min_buffer, (Npp32f *)&min_value,nppStream));
}
void GpuImage::MinAndCoords()
{
	MyAssertTrue(is_in_memory_gpu, "Memory not allocated");
	MyAssertTrue(is_in_real_space, "Not in real space");


	NppInit();
	BufferInit(b_minIDX);
	checkNppErrors(nppiMinIndx_32f_C1R_Ctx((const Npp32f *)real_values_gpu, pitch, npp_ROI, minIDX_buffer, (Npp32f *)&min_value, &min_idx.x, &min_idx.y,nppStream));
}
void GpuImage::Max()
{
	MyAssertTrue(is_in_memory_gpu, "Memory not allocated");
	MyAssertTrue(is_in_real_space, "Not in real space");


	NppInit();
	BufferInit(b_max);
	checkNppErrors(nppiMax_32f_C1R_Ctx((const Npp32f*)real_values_gpu, pitch, npp_ROI, max_buffer, (Npp32f *)&max_value,nppStream));
}
void GpuImage::MaxAndCoords()
{
	MyAssertTrue(is_in_memory_gpu, "Memory not allocated");
	MyAssertTrue(is_in_real_space, "Not in real space");


	NppInit();
	BufferInit(b_maxIDX);
	checkNppErrors(nppiMaxIndx_32f_C1R_Ctx((const Npp32f*)real_values_gpu, pitch, npp_ROI, maxIDX_buffer, (Npp32f *)&max_value, &max_idx.x, &max_idx.y, nppStream));
}
void GpuImage::MinMax()
{
	MyAssertTrue(is_in_memory_gpu, "Memory not allocated");
	MyAssertTrue(is_in_real_space, "Not in real space");


	NppInit();
	BufferInit(b_minmax);
	checkNppErrors(nppiMinMax_32f_C1R_Ctx((const Npp32f*)real_values_gpu, pitch, npp_ROI, (Npp32f *)&min_value, (Npp32f *)&max_value, minmax_buffer,nppStream));
}
void GpuImage::MinMaxAndCoords()
{
	MyAssertTrue(is_in_memory_gpu, "Memory not allocated");
	MyAssertTrue(is_in_real_space, "Not in real space");


	NppInit();
	BufferInit(b_minmaxIDX);
	checkNppErrors(nppiMinMaxIndx_32f_C1R_Ctx((const Npp32f*)real_values_gpu, pitch, npp_ROI, (Npp32f *)&min_value, (Npp32f *)&max_value,  &min_idx, &max_idx,minmax_buffer,nppStream));
}

void GpuImage::Mean()
{
	MyAssertTrue(is_in_memory_gpu, "Memory not allocated");
	MyAssertTrue(is_in_real_space, "Not in real space");

	NppInit();
	BufferInit(b_mean);
	checkNppErrors(nppiMean_32f_C1R_Ctx((const Npp32f*)real_values_gpu, pitch, npp_ROI, mean_buffer, npp_mean, nppStream));
	this->img_mean   = (float)*npp_mean;

}

void GpuImage::MeanStdDev()
{
	MyAssertTrue(is_in_memory_gpu, "Memory not allocated");
	MyAssertTrue(is_in_real_space, "Not in real space");


	NppInit();
	BufferInit(b_meanstddev);
	checkNppErrors(nppiMean_StdDev_32f_C1R_Ctx((const Npp32f*)real_values_gpu, pitch, npp_ROI, meanstddev_buffer, npp_mean, npp_stdDev,nppStream));
	this->img_mean   = (float)*npp_mean;
	this->img_stdDev = (float)*npp_stdDev;
}

void GpuImage::MultiplyPixelWise(GpuImage &other_image)
{
	MyAssertTrue(is_in_memory_gpu, "Memory not allocated");

	NppInit();
	checkNppErrors(nppiMul_32f_C1IR_Ctx((Npp32f*)other_image.real_values_gpu, pitch, (Npp32f*)real_values_gpu, pitch, npp_ROI,nppStream));
}


void GpuImage::AddConstant(const float add_val)
{
	MyAssertTrue(is_in_memory_gpu, "Memory not allocated");
	MyAssertTrue(is_in_real_space, "Not in real space");


	NppInit();
	checkNppErrors(nppiAddC_32f_C1IR_Ctx((Npp32f)add_val, (Npp32f*)real_values_gpu, pitch, npp_ROI,nppStream));
}

void GpuImage::SquareRealValues()
{
	MyAssertTrue(is_in_memory_gpu, "Memory not allocated");
	MyAssertTrue(is_in_real_space, "Not in real space");


	NppInit();
	checkNppErrors(nppiSqr_32f_C1IR_Ctx((Npp32f *)real_values_gpu, pitch, npp_ROI, nppStream));
}

void GpuImage::SquareRootRealValues()
{
	MyAssertTrue(is_in_memory_gpu, "Memory not allocated");
	MyAssertTrue(is_in_real_space, "Not in real space");


	NppInit();
	checkNppErrors(nppiSqrt_32f_C1IR_Ctx((Npp32f *)real_values_gpu, pitch, npp_ROI, nppStream));
}

void GpuImage::LogarithmRealValues()
{
  MyAssertTrue(is_in_memory_gpu, "Memory not allocated");
  
  NppInit();
  checkNppErrors(nppiLn_32f_C1IR_Ctx((Npp32f *)real_values_gpu, pitch, npp_ROI,nppStream));

}

void GpuImage::ExponentiateRealValues()
{
	MyAssertTrue(is_in_memory_gpu, "Memory not allocated");
	MyAssertTrue(is_in_real_space, "Not in real space");


	NppInit();
	checkNppErrors(nppiExp_32f_C1IR_Ctx((Npp32f *)real_values_gpu, pitch, npp_ROI, nppStream));
}

void GpuImage::CountInRange(float lower_bound, float upper_bound)
{
	MyAssertTrue(is_in_memory_gpu, "Memory not allocated");
	MyAssertTrue(is_in_real_space, "Not in real space");


	NppInit();
	checkNppErrors(nppiCountInRange_32f_C1R_Ctx((const Npp32f *)real_values_gpu, pitch, npp_ROI, &number_of_pixels_in_range,
											(Npp32f)lower_bound,(Npp32f)upper_bound,countinrange_buffer, nppStream));

}

float GpuImage::ReturnSumOfRealValues()
{
	// FIXME assuming padded values are zero
	MyAssertTrue(is_in_memory_gpu, "Memory not allocated");
	MyAssertTrue(is_in_real_space, "Not in real space");


	Npp64f sum_val;

	NppInit();
	BufferInit(b_sum);
	checkNppErrors(nppiSum_32f_C1R_Ctx((const Npp32f*)real_values_gpu, pitch, npp_ROI,sum_buffer,&sum_val, nppStream));

	return (float)sum_val;
}
void GpuImage::AddImage(GpuImage &other_image)
{
  // Add the real_values_gpu into a double array
	MyAssertTrue(HasSameDimensionsAs(&other_image), "Images have different dimensions");


	NppInit();
	checkNppErrors(nppiAdd_32f_C1IR_Ctx((const Npp32f*)other_image.real_values_gpu, pitch, (Npp32f*)real_values_gpu, pitch, npp_ROI, nppStream));

} 

void GpuImage::AddSquaredImage(GpuImage &other_image)
{
	// Add the real_values_gpu into a double array
	MyAssertTrue(HasSameDimensionsAs(&other_image), "Images have different dimensions");

	NppInit();
	checkNppErrors(nppiAddSquare_32f_C1IR_Ctx((const Npp32f*)other_image.real_values_gpu,  pitch, (Npp32f*)real_values_gpu,  pitch, npp_ROI, nppStream));
} 

void GpuImage::MultiplyByConstant(float scale_factor)
{
	MyAssertTrue(is_in_memory_gpu, "Memory not allocated");

	NppInit();
	checkNppErrors(nppiMulC_32f_C1IR_Ctx((Npp32f) scale_factor, (Npp32f*)real_values_gpu,  pitch, npp_ROI, nppStream));
}

void GpuImage::Conj()
{
	MyAssertTrue(is_in_memory_gpu, "Memory not allocated");
	MyAssertFalse(is_in_real_space, "Conj only supports complex images");

	float scale_factor = -1.0f;
	NppInit();
	checkNppErrors(nppiMulC_32f_C1IR_Ctx((Npp32f) (scale_factor+1), (Npp32f*)real_values_gpu,  dims.w/2*sizeof(float), npp_ROI,nppStream));
	// FIXME make sure that a) there isn't already a function fo rthis, b) you aren't striding out of bounds (mask instead_;
}

void GpuImage::Zeros()
{
 
  MyAssertTrue(is_meta_data_initialized, "Host meta data has not been copied");

  if ( ! is_in_memory_gpu )
  {
    checkCudaErrors(hipMalloc(&real_values_gpu, real_memory_allocated*sizeof(float)));
    complex_values_gpu = (hipfftComplex *)real_values_gpu;
    is_in_memory_gpu = true;
  }

  pre_checkErrorsAndTimingWithSynchronization(hipStreamPerThread);
  checkCudaErrors(hipMemsetAsync(real_values_gpu, 0, real_memory_allocated*sizeof(float), hipStreamPerThread));
  checkErrorsAndTimingWithSynchronization(hipStreamPerThread);

}


void GpuImage::CopyHostToDevice()
{
 
	MyAssertTrue(is_in_memory, "Host memory not allocated");

	if ( ! is_in_memory_gpu )
	{
		checkCudaErrors(hipMalloc(&real_values_gpu, real_memory_allocated*sizeof(float)));
		complex_values_gpu = (hipfftComplex *)real_values_gpu;
		is_in_memory_gpu = true;
	}

	pre_checkErrorsAndTimingWithSynchronization(hipStreamPerThread);
	checkCudaErrors(hipMemcpyAsync( real_values_gpu, pinnedPtr, real_memory_allocated*sizeof(float),hipMemcpyHostToDevice,hipStreamPerThread));
	checkErrorsAndTimingWithSynchronization(hipStreamPerThread);

	UpdateCpuFlags();

}

void GpuImage::CopyDeviceToHost(bool free_gpu_memory, bool unpin_host_memory)
{
 
	MyAssertTrue(is_in_memory_gpu, "GPU memory not allocated");
	// TODO other asserts on size etc.
	pre_checkErrorsAndTimingWithSynchronization(hipStreamPerThread);
	checkCudaErrors(hipMemcpyAsync(pinnedPtr, real_values_gpu, real_memory_allocated*sizeof(float),hipMemcpyDeviceToHost,hipStreamPerThread));
	checkErrorsAndTimingWithSynchronization(hipStreamPerThread);
	//  checkCudaErrors(hipMemcpyAsync(real_values, real_values_gpu, real_memory_allocated*sizeof(float),hipMemcpyDeviceToHost,hipStreamPerThread));
	// TODO add asserts etc.
	if (free_gpu_memory) { hipFree(real_values_gpu) ; } // FIXME what about the other structures
	if (unpin_host_memory && is_host_memory_pinned)
	{
		hipHostUnregister(real_values);
		is_host_memory_pinned = false;
	}

}

void GpuImage::CopyDeviceToHost(Image &cpu_image, bool should_block_until_complete, bool free_gpu_memory)
{

	MyAssertTrue(is_in_memory_gpu, "GPU memory not allocated");
	// TODO other asserts on size etc.


	float* tmpPinnedPtr;
	// FIXME for now always pin the memory - this might be a bad choice for single copy or small images, but is required for asynch xfer and is ~2x as fast after pinning
	hipHostRegister(cpu_image.real_values, sizeof(float)*real_memory_allocated, hipHostRegisterDefault);
	hipHostGetDevicePointer( &tmpPinnedPtr, cpu_image.real_values, 0);

	pre_checkErrorsAndTimingWithSynchronization(hipStreamPerThread);
	checkCudaErrors(hipMemcpyAsync(tmpPinnedPtr, real_values_gpu, real_memory_allocated*sizeof(float),hipMemcpyDeviceToHost,hipStreamPerThread));
	checkErrorsAndTimingWithSynchronization(hipStreamPerThread);

	if (should_block_until_complete) checkCudaErrors(hipStreamSynchronize(hipStreamPerThread));
	// TODO add asserts etc.
	if (free_gpu_memory) { hipFree(real_values_gpu) ; } // FIXME what about the other structures

	hipHostUnregister(tmpPinnedPtr);


}

void GpuImage::CopyVolumeHostToDevice()
{


  // FIXME not working
    bool is_working = false;
    MyAssertTrue(is_working, "CopyVolumeHostToDevice is not properly worked out");

		d_pitchedPtr = { 0 };
		d_extent = make_hipExtent(dims.x * sizeof(float), dims.y, dims.z);
		checkCudaErrors(hipMalloc3D(&d_pitchedPtr, d_extent)); // Complex values need to be pointed
    this->real_values_gpu = (hipfftReal *)d_pitchedPtr.ptr; // Set the values here

		d_3dparams        = { 0 };
		d_3dparams.srcPtr = make_hipPitchedPtr((void*)real_values, dims.x * sizeof(float), dims.x, dims.y);
		d_3dparams.dstPtr = d_pitchedPtr;
		d_3dparams.extent = d_extent;
		d_3dparams.kind   = hipMemcpyHostToDevice;
		checkCudaErrors(hipMemcpy3D(&d_3dparams));

}

void GpuImage::CopyVolumeDeviceToHost(bool free_gpu_memory, bool unpin_host_memory)
{

  // FIXME not working
    bool is_working = false;
    MyAssertTrue(is_working, "CopyVolumeDeviceToHost is not properly worked out");

    if ( ! is_in_memory )
    {
		  checkCudaErrors(hipHostMalloc(&real_values, real_memory_allocated*sizeof(float)));
    }
    h_pitchedPtr = make_hipPitchedPtr((void*)real_values, dims.x * sizeof(float), dims.x, dims.y);
		h_extent = make_hipExtent(dims.x * sizeof(float), dims.y, dims.z);
		h_3dparams        = { 0 };
		h_3dparams.srcPtr = d_pitchedPtr;
		h_3dparams.dstPtr = h_pitchedPtr;
		h_3dparams.extent = h_extent;
		h_3dparams.kind   = hipMemcpyDeviceToHost;
		checkCudaErrors(hipMemcpy3D(&h_3dparams));

    is_in_memory = true;

    // TODO add asserts etc.
    if (free_gpu_memory) 
    { hipFree(d_pitchedPtr.ptr) ; } // FIXME what about the other structures
    if (unpin_host_memory && is_host_memory_pinned) 
    {
      hipHostUnregister(real_values);
      is_host_memory_pinned = false;
    }
}

void GpuImage::ForwardFFT(bool should_scale)
{

	bool is_half_precision = false;

	MyAssertTrue(is_in_memory_gpu, "Gpu memory not allocated");
	MyAssertTrue(is_in_real_space, "Image alread in Fourier space");

	if ( ! is_fft_planned )
	{
		SetCufftPlan();
	}

	// For reference to clear hipfftXtClearCallback(hipfftHandle lan, hipfftXtCallbackType type);
	if ( is_half_precision && ! is_set_convertInputf16Tof32 )
	{
		hipfftCallbackLoadR h_ConvertInputf16Tof32Ptr;
		checkCudaErrors(hipMemcpyFromSymbol(&h_ConvertInputf16Tof32Ptr,HIP_SYMBOL(d_ConvertInputf16Tof32Ptr), sizeof(h_ConvertInputf16Tof32Ptr)));
		checkCudaErrors(hipfftXtSetCallback(cuda_plan_forward, (void **)&h_ConvertInputf16Tof32Ptr, HIPFFT_CB_LD_REAL, 0));
		is_set_convertInputf16Tof32 = true;
		//	  checkCudaErrors(hipFree(norm_factor));
		//	  this->MultiplyByConstant(ft_normalization_factor*ft_normalization_factor);
	}
	if (should_scale)
	{
		this->MultiplyByConstant(ft_normalization_factor*ft_normalization_factor);
	}

//	if (should_scale && ! is_set_scaleFFTAndStore)
//	{
//
//		float ft_norm_sq = ft_normalization_factor*ft_normalization_factor;
//		checkCudaErrors(hipMalloc((void **)&d_scale_factor, sizeof(float)));
//		checkCudaErrors(hipMemcpyAsync(d_scale_factor, &ft_norm_sq, sizeof(float), hipMemcpyHostToDevice, hipStreamPerThread));
//		checkCudaErrors(hipStreamSynchronize(hipStreamPerThread));
//
//		hipfftCallbackStoreC h_scaleFFTAndStorePtr;
//		checkCudaErrors(hipMemcpyFromSymbol(&h_scaleFFTAndStorePtr,HIP_SYMBOL(d_scaleFFTAndStorePtr), sizeof(h_scaleFFTAndStorePtr)));
//		checkCudaErrors(hipfftXtSetCallback(cuda_plan_forward, (void **)&h_scaleFFTAndStorePtr, HIPFFT_CB_ST_COMPLEX, (void **)&d_scale_factor));
//		is_set_scaleFFTAndStore = true;
//	}


//	BufferInit(b_image);
//    checkCudaErrors(hipfftExecR2C(this->cuda_plan_forward, (hipfftReal*)real_values_gpu, (hipfftComplex*)image_buffer->complex_values));

    checkCudaErrors(hipfftExecR2C(this->cuda_plan_forward, (hipfftReal*)real_values_gpu, (hipfftComplex*)complex_values_gpu));

    is_in_real_space = false;


}

void GpuImage::ForwardFFTAndClipInto(GpuImage &image_to_insert, bool should_scale)
{

	MyAssertTrue(is_in_memory_gpu, "Gpu memory not allocated");
	MyAssertTrue(image_to_insert.is_in_memory_gpu, "Gpu memory in image to insert not allocated");
	MyAssertTrue(is_in_real_space, "Image alread in Fourier space");
	MyAssertTrue(image_to_insert.is_in_real_space, "I in image to insert alread in Fourier space");


	if ( ! is_fft_planned )
	{
		SetCufftPlan();
	}

	// For reference to clear hipfftXtClearCallback(hipfftHandle lan, hipfftXtCallbackType type);
	if ( ! is_set_realLoadAndClipInto )
	{
		hipfftCallbackLoadR h_realLoadAndClipInto;
		CB_realLoadAndClipInto_params* d_params;
		CB_realLoadAndClipInto_params h_params;


		h_params.target = (hipfftReal *)image_to_insert.real_values_gpu;
		h_params.i_lower_bound =  this->physical_address_of_box_center.x - image_to_insert.physical_address_of_box_center.x;
		h_params.j_lower_bound =  this->physical_address_of_box_center.y - image_to_insert.physical_address_of_box_center.y;
		h_params.i_upper_bound = h_params.i_lower_bound + image_to_insert.dims.x - 1;
		h_params.j_upper_bound = h_params.j_lower_bound + image_to_insert.dims.y - 1;
		// For the offset in memory, we need to consider the physical dimensions of the images and not just the logical.
		h_params.nx_small = (image_to_insert.dims.w);
		h_params.nx_large 	  = this->dims.w;

		checkCudaErrors(hipMalloc((void **)&d_params,sizeof(CB_realLoadAndClipInto_params)));
		checkCudaErrors(hipMemcpyAsync(d_params, &h_params, sizeof(CB_realLoadAndClipInto_params), hipMemcpyHostToDevice, hipStreamPerThread));
		checkCudaErrors(hipMemcpyFromSymbol(&h_realLoadAndClipInto,HIP_SYMBOL(d_realLoadAndClipInto), sizeof(h_realLoadAndClipInto)));
		checkCudaErrors(hipStreamSynchronize(hipStreamPerThread));

		checkCudaErrors(hipfftXtSetCallback(cuda_plan_forward, (void **)&h_realLoadAndClipInto, HIPFFT_CB_LD_REAL, (void **)&d_params));
		is_set_realLoadAndClipInto = true;


		//	  checkCudaErrors(hipFree(norm_factor));
		//	  this->MultiplyByConstant(ft_normalization_factor*ft_normalization_factor);
	}
	if (should_scale)
	{
		this->MultiplyByConstant(ft_normalization_factor*ft_normalization_factor);
	}




//	BufferInit(b_image);
//    checkCudaErrors(hipfftExecR2C(this->cuda_plan_forward, (hipfftReal*)real_values_gpu, (hipfftComplex*)image_buffer->complex_values));

    checkCudaErrors(hipfftExecR2C(this->cuda_plan_forward, (hipfftReal*)real_values_gpu, (hipfftComplex*)complex_values_gpu));

    is_in_real_space = false;


}

void GpuImage::BackwardFFT()
{

	MyAssertTrue(is_in_memory_gpu, "Gpu memory not allocated");
	MyAssertFalse(is_in_real_space, "Image is already in real space");

	if ( ! is_fft_planned )
	{
		SetCufftPlan();
	}

	//  BufferInit(b_image);
	//  checkCudaErrors(hipfftExecC2R(this->cuda_plan_inverse, (hipfftComplex*)image_buffer->complex_values, (hipfftReal*)real_values_gpu));

	checkCudaErrors(hipfftExecC2R(this->cuda_plan_inverse, (hipfftComplex*)complex_values_gpu, (hipfftReal*)real_values_gpu));

	is_in_real_space = true;

}

template < typename T > void GpuImage::BackwardFFTAfterComplexConjMul(T* image_to_multiply, bool load_half_precision)
{

	MyAssertTrue(is_in_memory_gpu, "Gpu memory not allocated");
	MyAssertFalse(is_in_real_space, "Image is already in real space");

	if ( ! is_fft_planned )
	{
		SetCufftPlan();
	}

	if ( ! is_set_complexConjMulLoad )
	{
		hipfftCallbackStoreC h_complexConjMulLoad;


			CB_complexConjMulLoad_params<T>* d_params;
			CB_complexConjMulLoad_params<T> h_params;
			h_params.scale = ft_normalization_factor*ft_normalization_factor;
			h_params.target = (T *)image_to_multiply;
			checkCudaErrors(hipMalloc((void **)&d_params,sizeof(CB_complexConjMulLoad_params<T>)));
			checkCudaErrors(hipMemcpyAsync(d_params, &h_params, sizeof(CB_complexConjMulLoad_params<T>), hipMemcpyHostToDevice, hipStreamPerThread));

			if (load_half_precision)
			{
				checkCudaErrors(hipMemcpyFromSymbol(&h_complexConjMulLoad,HIP_SYMBOL(d_complexConjMulLoad_16f), sizeof(h_complexConjMulLoad)));
			}
			else
			{
				checkCudaErrors(hipMemcpyFromSymbol(&h_complexConjMulLoad,HIP_SYMBOL(d_complexConjMulLoad_32f), sizeof(h_complexConjMulLoad)));
			}

			checkCudaErrors(hipStreamSynchronize(hipStreamPerThread));
			checkCudaErrors(hipfftXtSetCallback(cuda_plan_inverse, (void **)&h_complexConjMulLoad, HIPFFT_CB_LD_COMPLEX, (void **)&d_params));

//		d_complexConjMulLoad;
		is_set_complexConjMulLoad = true;
	}

	//  BufferInit(b_image);
	//  checkCudaErrors(hipfftExecC2R(this->cuda_plan_inverse, (hipfftComplex*)image_buffer->complex_values, (hipfftReal*)real_values_gpu));

	checkCudaErrors(hipfftExecC2R(this->cuda_plan_inverse, (hipfftComplex*)complex_values_gpu, (hipfftReal*)real_values_gpu));

	is_in_real_space = true;

}
template void GpuImage::BackwardFFTAfterComplexConjMul(__half2* image_to_multiply, bool load_half_precision);
template void GpuImage::BackwardFFTAfterComplexConjMul(hipfftComplex* image_to_multiply, bool load_half_precision);



void GpuImage::Wait()
{
  checkCudaErrors(hipStreamSynchronize(hipStreamPerThread));
}

void GpuImage::SwapRealSpaceQuadrants()
{

	MyAssertTrue(is_in_memory_gpu, "Gpu memory not allocated");

	bool must_fft = false;

	float x_shift_to_apply;
	float y_shift_to_apply;
	float z_shift_to_apply;

	if (is_in_real_space == true)
	{
		must_fft = true;
		ForwardFFT(true);
	}

	if (object_is_centred_in_box == true)
	{
		x_shift_to_apply = float(physical_address_of_box_center.x);
		y_shift_to_apply = float(physical_address_of_box_center.y);
		z_shift_to_apply = float(physical_address_of_box_center.z);
	}
	else
	{
		if (IsEven(dims.x) == true)
		{
			x_shift_to_apply = float(physical_address_of_box_center.x);
		}
		else
		{
			x_shift_to_apply = float(physical_address_of_box_center.x) - 1.0;
		}

		if (IsEven(dims.y) == true)
		{
			y_shift_to_apply = float(physical_address_of_box_center.y);
		}
		else
		{
			y_shift_to_apply = float(physical_address_of_box_center.y) - 1.0;
		}

		if (IsEven(dims.z) == true)
		{
			z_shift_to_apply = float(physical_address_of_box_center.z);
		}
		else
		{
			z_shift_to_apply = float(physical_address_of_box_center.z) - 1.0;
		}
	}


	if (dims.z == 1)
	{
		z_shift_to_apply = 0.0;
	}

	PhaseShift(x_shift_to_apply, y_shift_to_apply, z_shift_to_apply);

	if (must_fft == true) BackwardFFT();


	// keep track of center;
	if (object_is_centred_in_box == true) object_is_centred_in_box = false;
	else object_is_centred_in_box = true;
}




void GpuImage::PhaseShift(float wanted_x_shift, float wanted_y_shift, float wanted_z_shift)
{

	MyAssertTrue(is_in_memory_gpu, "Gpu memory not allocated");

	bool need_to_fft = false;
	if (is_in_real_space == true)
	{
    wxPrintf("Doing forward fft in phase shift function\n\n");
		ForwardFFT(true);
		need_to_fft = true;
	}

  float3 shifts = make_float3(wanted_x_shift, wanted_y_shift, wanted_z_shift);
  // TODO set the TPB and inline function for grid

  
  dim3 threadsPerBlock(32, 32, 1);
  dim3 gridDims((dims.w/2 + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                (dims.y + threadsPerBlock.y - 1) / threadsPerBlock.y, dims.z); 

	pre_checkErrorsAndTimingWithSynchronization(hipStreamPerThread);
	PhaseShiftKernel<< <gridDims, threadsPerBlock,0,hipStreamPerThread>> >(complex_values_gpu, 
                                                        dims, shifts,
                                                        physical_address_of_box_center,
                                                        physical_index_of_first_negative_frequency,
                                                        physical_upper_bound_complex);
  
    checkErrorsAndTimingWithSynchronization(hipStreamPerThread);

	if (need_to_fft == true) BackwardFFT();

}

__device__ __forceinline__ float
d_ReturnPhaseFromShift(float real_space_shift, float distance_from_origin, float dimension_size)
{
	return real_space_shift * distance_from_origin * 2.0 * PI / dimension_size;
}

__device__ __forceinline__ void
d_Return3DPhaseFromIndividualDimensions( float phase_x, float phase_y, float phase_z, float2 &angles)
{
	float temp_phase = -phase_x-phase_y-phase_z;
	__sincosf(temp_phase, &angles.y, &angles.x); // To use as cos.x + i*sin.y
}


 __device__ __forceinline__ int
d_ReturnFourierLogicalCoordGivenPhysicalCoord_X(int physical_index, 
                                                int logical_x_dimension,
                                                int physical_address_of_box_center_x)
{
//	MyAssertTrue(is_in_memory, "Memory not allocated");
//	MyAssertTrue(physical_index <= physical_upper_bound_complex_x, "index out of bounds");

    //if (physical_index >= physical_index_of_first_negative_frequency_x)
    if (physical_index > physical_address_of_box_center_x)
    {
    	 return physical_index - logical_x_dimension;
    }
    else return physical_index;
}


 __device__ __forceinline__ int
d_ReturnFourierLogicalCoordGivenPhysicalCoord_Y(int physical_index,
                                                int logical_y_dimension,
                                                int physical_index_of_first_negative_frequency_y )
{
//	MyAssertTrue(is_in_memory, "Memory not allocated");
//	MyAssertTrue(physical_index <= physical_upper_bound_complex_y, "index out of bounds");

    if (physical_index >= physical_index_of_first_negative_frequency_y)
    {
    	 return physical_index - logical_y_dimension;
    }
    else return physical_index;
}


 __device__ __forceinline__ int
d_ReturnFourierLogicalCoordGivenPhysicalCoord_Z(int physical_index,

                                                int logical_z_dimension,
                                                int physical_index_of_first_negative_frequency_z )
{
//	MyAssertTrue(is_in_memory, "Memory not allocated");
//	MyAssertTrue(physical_index <= physical_upper_bound_complex_z, "index out of bounds");

    if (physical_index >= physical_index_of_first_negative_frequency_z)
    {
    	 return physical_index - logical_z_dimension;
    }
    else return physical_index;
}

 __device__ __forceinline__ int
d_ReturnReal1DAddressFromPhysicalCoord(int3 coords, int4 img_dims)
{
	return ( (((int)coords.z*(int)img_dims.y + coords.y) * (int)img_dims.w)  + (int)coords.x) ;
}

 __device__ __forceinline__ int
d_ReturnFourier1DAddressFromPhysicalCoord(int3 wanted_dims, int3 physical_upper_bound_complex)
{
	return ( (int)((physical_upper_bound_complex.y + 1) * wanted_dims.z + wanted_dims.y) *
            (int)(physical_upper_bound_complex.x + 1) + (int)wanted_dims.x );
}


__global__ void ClipIntoRealKernel(hipfftReal* real_values_gpu,
                                   hipfftReal* other_image_real_values_gpu,
                                   int4 dims, 
                                   int4 other_dims,
                                   int3 physical_address_of_box_center, 
                                   int3 other_physical_address_of_box_center,
                                   int3 wanted_coordinate_of_box_center, 
                                   float wanted_padding_value)
{
  int3 other_coord = make_int3(blockIdx.x*blockDim.x + threadIdx.x,
                               blockIdx.y*blockDim.y + threadIdx.y,
                               blockIdx.z);

  int3 coord = make_int3(0, 0, 0); 
  
  if (other_coord.x < other_dims.x &&
      other_coord.y < other_dims.y &&
      other_coord.z < other_dims.z)
  {

    coord.z = physical_address_of_box_center.z + wanted_coordinate_of_box_center.z + 
              other_coord.z - other_physical_address_of_box_center.z;

    coord.y = physical_address_of_box_center.y + wanted_coordinate_of_box_center.y + 
              other_coord.y - other_physical_address_of_box_center.y;

    coord.x = physical_address_of_box_center.x + wanted_coordinate_of_box_center.x + 
              other_coord.x - other_physical_address_of_box_center.x;

    if (coord.z < 0 || coord.z >= dims.z || 
        coord.y < 0 || coord.y >= dims.y ||
        coord.x < 0 || coord.x >= dims.x)
    {
      other_image_real_values_gpu[ d_ReturnReal1DAddressFromPhysicalCoord(other_coord, other_dims) ] = wanted_padding_value;
    }
    else
    {
      other_image_real_values_gpu[ d_ReturnReal1DAddressFromPhysicalCoord(other_coord, other_dims) ] = 
                  real_values_gpu[ d_ReturnReal1DAddressFromPhysicalCoord(coord, dims) ];
    }





  }
//		for (kk = 0; kk < other_image->logical_z_dimension; kk++)
//		{
//			kk_logi = kk - other_image->physical_address_of_box_center_z;
//			k = physical_address_of_box_center_z + wanted_coordinate_of_box_center_z + kk_logi;

//			for (jj = 0; jj < other_image->logical_y_dimension; jj++)
//			{
//				jj_logi = jj - other_image->physical_address_of_box_center_y;
//				j = physical_address_of_box_center_y + wanted_coordinate_of_box_center_y + jj_logi;

//				for (ii = 0; ii < other_image->logical_x_dimension; ii++)
//				{
//					ii_logi = ii - other_image->physical_address_of_box_center_x;
//					i = physical_address_of_box_center_x + wanted_coordinate_of_box_center_x + ii_logi;

//					if (k < 0 || k >= logical_z_dimension || j < 0 || j >= logical_y_dimension || i < 0 || i >= logical_x_dimension)
//					{
//						other_image->real_values[pixel_counter] = wanted_padding_value;
//					}
//					else
//					{
//						other_image->real_values[pixel_counter] = ReturnRealPixelFromPhysicalCoord(i, j, k);
//					}

//					pixel_counter++;
//				}

//				pixel_counter+=other_image->padding_jump_value;
//			}
//		}
//	}

}
__global__ void PhaseShiftKernel(hipfftComplex* d_input, 
                                 int4 dims, float3 shifts, 
                                 int3 physical_address_of_box_center, 
                                 int3 physical_index_of_first_negative_frequency,
                                 int3 physical_upper_bound_complex)
{
	
// FIXME it probably makes sense so just just a linear grid launch and save the extra indexing
  int3 wanted_dims = make_int3(blockIdx.x*blockDim.x + threadIdx.x,
                               blockIdx.y*blockDim.y + threadIdx.y,
                               blockIdx.z);

  float2 init_vals;
  float2 angles;

// FIXME This should probably use cuBlas
  if (wanted_dims.x <= physical_upper_bound_complex.x && 
      wanted_dims.y <= physical_upper_bound_complex.y && 
      wanted_dims.z <= physical_upper_bound_complex.z)
  {
  
    
    d_Return3DPhaseFromIndividualDimensions( d_ReturnPhaseFromShift(
                                              shifts.x, 
                                              wanted_dims.x,
                                              dims.x), 
                                             d_ReturnPhaseFromShift(
                                              shifts.y, 
                                                d_ReturnFourierLogicalCoordGivenPhysicalCoord_Y(
                                                  wanted_dims.y,
                                                  dims.y,
                                                  physical_index_of_first_negative_frequency.y), 
                                              dims.y),
                                             d_ReturnPhaseFromShift(
                                              shifts.z, 
                                              d_ReturnFourierLogicalCoordGivenPhysicalCoord_Z(
                                                wanted_dims.z,
                                                dims.z,
                                                physical_index_of_first_negative_frequency.z), 
                                              dims.z),
                                             angles);
    
    int address = d_ReturnFourier1DAddressFromPhysicalCoord(wanted_dims, physical_upper_bound_complex);
    init_vals.x = d_input[ address ].x;
    init_vals.y = d_input[ address ].y;
    d_input[ address ].x = init_vals.x*angles.x - init_vals.y*angles.y;
    d_input[ address ].y = init_vals.x*angles.y + init_vals.y*angles.x;
  }
  

}


// If you don't want to clip from the center, you can give wanted_coordinate_of_box_center_{x,y,z}. This will define the pixel in the image at which other_image will be centered. (0,0,0) means center of image. This is a dumbed down version that does not fill with noise.
void GpuImage::ClipInto(GpuImage *other_image, float wanted_padding_value,                 
                        bool fill_with_noise, float wanted_noise_sigma,
                        int wanted_coordinate_of_box_center_x, 
                        int wanted_coordinate_of_box_center_y, 
                        int wanted_coordinate_of_box_center_z)
{

	MyAssertTrue(is_in_memory_gpu, "Memory not allocated");
	MyAssertTrue(other_image->is_in_memory_gpu, "Other image Memory not allocated");
	MyAssertTrue(is_in_real_space, "Clip into is only set up for real space on the gpu currently");


  int3 wanted_coordinate_of_box_center = make_int3(wanted_coordinate_of_box_center_x, 
                                                   wanted_coordinate_of_box_center_y, 
                                                   wanted_coordinate_of_box_center_z);

	other_image->is_in_real_space = is_in_real_space;
	other_image->object_is_centred_in_box = object_is_centred_in_box;

	if (is_in_real_space == true)
	{

		MyAssertTrue(object_is_centred_in_box, "real space image, not centred in box");

    ReturnLaunchParamters(other_image->dims, true);

	pre_checkErrorsAndTimingWithSynchronization(hipStreamPerThread);
    ClipIntoRealKernel<< <gridDims, threadsPerBlock, 0, hipStreamPerThread>> >(real_values_gpu,
                                                              other_image->real_values_gpu,
                                                              dims, 
                                                              other_image->dims,
                                                              physical_address_of_box_center,
                                                              other_image->physical_address_of_box_center, 
                                                              wanted_coordinate_of_box_center, 
                                                              wanted_padding_value);
    checkErrorsAndTimingWithSynchronization(hipStreamPerThread);
  }
	else
  {
    // FIXME
    wxPrintf("\n\nClipInto is only setup for real space!\n\n");
    exit(-1);
  }
//	{
//		for (kk = 0; kk <= other_image->physical_upper_bound_complex_z; kk++)
//		{
//			temp_logical_z = other_image->ReturnFourierLogicalCoordGivenPhysicalCoord_Z(kk);

//			//if (temp_logical_z > logical_upper_bound_complex_z || temp_logical_z < logical_lower_bound_complex_z) continue;

//			for (jj = 0; jj <= other_image->physical_upper_bound_complex_y; jj++)
//			{
//				temp_logical_y = other_image->ReturnFourierLogicalCoordGivenPhysicalCoord_Y(jj);

//				//if (temp_logical_y > logical_upper_bound_complex_y || temp_logical_y < logical_lower_bound_complex_y) continue;

//				for (ii = 0; ii <= other_image->physical_upper_bound_complex_x; ii++)
//				{
//					temp_logical_x = ii;

//					//if (temp_logical_x > logical_upper_bound_complex_x || temp_logical_x < logical_lower_bound_complex_x) continue;

//					if (fill_with_noise == false) other_image->complex_values[pixel_counter] = ReturnComplexPixelFromLogicalCoord(temp_logical_x, temp_logical_y, temp_logical_z, wanted_padding_value + I * 0.0f);
//					else
//					{

//						if (temp_logical_x < logical_lower_bound_complex_x || temp_logical_x > logical_upper_bound_complex_x || temp_logical_y < logical_lower_bound_complex_y ||temp_logical_y > logical_upper_bound_complex_y || temp_logical_z < logical_lower_bound_complex_z || temp_logical_z > logical_upper_bound_complex_z)
//						{
//							other_image->complex_values[pixel_counter] = (global_random_number_generator.GetNormalRandom() * wanted_noise_sigma) + (I * global_random_number_generator.GetNormalRandom() * wanted_noise_sigma);
//						}
//						else
//						{
//							other_image->complex_values[pixel_counter] = complex_values[ReturnFourier1DAddressFromLogicalCoord(temp_logical_x,temp_logical_y, temp_logical_z)];

//						}


//					}
//					pixel_counter++;

//				}

//			}
//		}


//		// When we are clipping into a larger volume in Fourier space, there is a half-plane (vol) or half-line (2D image) at Nyquist for which FFTW
//		// does not explicitly tell us the values. We need to fill them in.
//		if (logical_y_dimension < other_image->logical_y_dimension || logical_z_dimension < other_image->logical_z_dimension)
//		{
//			// For a 2D image
//			if (logical_z_dimension == 1)
//			{
//				jj = physical_index_of_first_negative_frequency_y;
//				for (ii = 0; ii <= physical_upper_bound_complex_x; ii++)
//				{
//					other_image->complex_values[other_image->ReturnFourier1DAddressFromPhysicalCoord(ii,jj,0)] = complex_values[ReturnFourier1DAddressFromPhysicalCoord(ii,jj,0)];
//				}
//			}
//			// For a 3D volume
//			else
//			{

//				// Deal with the positive Nyquist of the 2nd dimension
//				for (kk_logi = logical_lower_bound_complex_z; kk_logi <= logical_upper_bound_complex_z; kk_logi ++)
//				{
//					jj = physical_index_of_first_negative_frequency_y;
//					jj_logi = logical_lower_bound_complex_y;
//					for (ii = 0; ii <= physical_upper_bound_complex_x; ii++)
//					{
//						other_image->complex_values[other_image->ReturnFourier1DAddressFromLogicalCoord(ii,jj,kk_logi)] = complex_values[ReturnFourier1DAddressFromLogicalCoord(ii,jj_logi,kk_logi)];
//					}
//				}


//				// Deal with the positive Nyquist in the 3rd dimension
//				kk = physical_index_of_first_negative_frequency_z;
//				int kk_mirror = other_image->logical_z_dimension - physical_index_of_first_negative_frequency_z;
//				//wxPrintf("\nkk = %i; kk_mirror = %i\n",kk,kk_mirror);
//				int jj_mirror;
//				//wxPrintf("Will loop jj from %i to %i\n",1,physical_index_of_first_negative_frequency_y);
//				for (jj = 1; jj <= physical_index_of_first_negative_frequency_y; jj ++ )
//				{
//					//jj_mirror = other_image->logical_y_dimension - jj;
//					jj_mirror = jj;
//					for (ii = 0; ii <= physical_upper_bound_complex_x; ii++ )
//					{
//						//wxPrintf("(1) ii = %i; jj = %i; kk = %i; jj_mirror = %i; kk_mirror = %i\n",ii,jj,kk,jj_mirror,kk_mirror);
//						other_image->complex_values[other_image-> ReturnFourier1DAddressFromPhysicalCoord(ii,jj,kk)] = other_image->complex_values[other_image->ReturnFourier1DAddressFromPhysicalCoord(ii,jj_mirror,kk_mirror)];
//					}
//				}
//				//wxPrintf("Will loop jj from %i to %i\n", other_image->logical_y_dimension - physical_index_of_first_negative_frequency_y, other_image->logical_y_dimension - 1);
//				for (jj = other_image->logical_y_dimension - physical_index_of_first_negative_frequency_y; jj <= other_image->logical_y_dimension - 1; jj ++)
//				{
//					//jj_mirror = other_image->logical_y_dimension - jj;
//					jj_mirror = jj;
//					for (ii = 0; ii <= physical_upper_bound_complex_x; ii++ )
//					{
//						//wxPrintf("(2) ii = %i; jj = %i; kk = %i; jj_mirror = %i; kk_mirror = %i\n",ii,jj,kk,jj_mirror,kk_mirror);
//						other_image->complex_values[other_image-> ReturnFourier1DAddressFromPhysicalCoord(ii,jj,kk)] = other_image->complex_values[other_image->ReturnFourier1DAddressFromPhysicalCoord(ii,jj_mirror,kk_mirror)];
//					}
//				}
//				jj = 0;
//				for (ii = 0; ii <= physical_upper_bound_complex_x; ii++)
//				{
//					other_image->complex_values[other_image->ReturnFourier1DAddressFromPhysicalCoord(ii,jj,kk)] = other_image->complex_values[other_image->ReturnFourier1DAddressFromPhysicalCoord(ii,jj,kk_mirror)];
//				}

//			}
//		}


//	}

}

void GpuImage::QuickAndDirtyWriteSlices(std::string filename, int first_slice, int last_slice)
{

	MyAssertTrue(is_in_memory_gpu, "Memory not allocated");
  Image buffer_img;
  buffer_img.Allocate(dims.x, dims.y, dims.z, true);

  buffer_img.is_in_real_space = is_in_real_space;
  buffer_img.object_is_centred_in_box = object_is_centred_in_box;
  // Implicitly waiting on work to finish since copy is queued in the stream
  checkCudaErrors(hipMemcpy((void*)buffer_img.real_values,(const void*)real_values_gpu, real_memory_allocated*sizeof(float),hipMemcpyDeviceToHost));
  bool OverWriteSlices = true;
  float pixelSize = 0.0f;

  buffer_img.QuickAndDirtyWriteSlices(filename, first_slice, last_slice, OverWriteSlices, pixelSize);
  buffer_img.Deallocate();
}

void GpuImage::SetCufftPlan(bool use_half_precision)
{

    int rank;
    long long int* fftDims;
    long long int* inembed;
    long long int* onembed;


    checkCudaErrors(hipfftCreate(&cuda_plan_forward));
    checkCudaErrors(hipfftCreate(&cuda_plan_inverse));

    checkCudaErrors(hipfftSetStream(cuda_plan_forward, hipStreamPerThread));
    checkCudaErrors(hipfftSetStream(cuda_plan_forward, hipStreamPerThread));

    if (dims.z > 1) 
    { 
      rank = 3;
      fftDims = new long long int[rank];
      inembed = new long long int[rank];
      onembed = new long long int[rank];
  
      fftDims[0] = dims.z;
      fftDims[1] = dims.y;
      fftDims[2] = dims.x;

      inembed[0] = dims.z;
      inembed[1] = dims.y;
      inembed[2] = dims.w; // Storage dimension (padded)

      onembed[0] = dims.z;
      onembed[1] = dims.y;
      onembed[2] = dims.w/2; // Storage dimension (padded)

      
    }
    else if (dims.y > 1) 
    { 
      wxPrintf("\n\nAllocating a 2d Plan\n\n");
      rank = 2;
      fftDims = new long long int[rank];
      inembed = new long long int[rank];
      onembed = new long long int[rank];

      fftDims[0] = dims.y;
      fftDims[1] = dims.x;

      inembed[0] = dims.y;
      inembed[1] = dims.w;

      onembed[0] = dims.y;
      onembed[1] = dims.w/2;

    }    
    else 
    { 
      rank = 1; 
      fftDims = new long long int[rank];
      inembed = new long long int[rank];
      onembed = new long long int[rank];
  
      fftDims[0] = dims.x;

      inembed[0] = dims.w;
      onembed[0] = dims.w/2;
    }



    int iBatch = 1;

// As far as I can tell, the padded layout must be assumed and onembed/inembed
// are not needed. TODO ask John about this.

    if (use_half_precision)
    {
    	checkCudaErrors(hipfftXtMakePlanMany(cuda_plan_forward, rank, fftDims,
    				  NULL, NULL, NULL, HIP_R_16F,
    				  NULL, NULL, NULL, HIP_C_16F, iBatch, &cuda_plan_worksize_forward, HIP_C_16F));
    	checkCudaErrors(hipfftXtMakePlanMany(cuda_plan_inverse, rank, fftDims,
    				  NULL, NULL, NULL, HIP_C_16F,
    				  NULL, NULL, NULL, HIP_R_16F, iBatch, &cuda_plan_worksize_inverse, HIP_R_16F));
    }
    else
    {
    	checkCudaErrors(hipfftXtMakePlanMany(cuda_plan_forward, rank, fftDims,
    				  NULL, NULL, NULL, HIP_R_32F,
    				  NULL, NULL, NULL, HIP_C_32F, iBatch, &cuda_plan_worksize_forward, HIP_C_32F));
    	checkCudaErrors(hipfftXtMakePlanMany(cuda_plan_inverse, rank, fftDims,
    				  NULL, NULL, NULL, HIP_C_32F,
    				  NULL, NULL, NULL, HIP_R_32F, iBatch, &cuda_plan_worksize_inverse, HIP_R_32F));
    }

//    hipfftPlanMany(&dims.cuda_plan_forward, rank, fftDims, 
//                  inembed, iStride, iDist, 
//                  onembed, oStride, oDist, HIPFFT_R2C, iBatch);
//    hipfftPlanMany(&dims.cuda_plan_inverse, rank, fftDims, 
//                  onembed, oStride, oDist, 
//                  inembed, iStride, iDist, HIPFFT_C2R, iBatch);

 

    delete [] fftDims;
    delete [] inembed;
    delete [] onembed;

	is_fft_planned = true;

  }  



void GpuImage::Deallocate()
{


  if (is_host_memory_pinned) hipHostUnregister(real_values);

  hipFree(tmpVal);
  hipFree(tmpValComplex);

  if (is_fft_planned)
  {
    checkCudaErrors(hipfftDestroy(cuda_plan_inverse));
    checkCudaErrors(hipfftDestroy(cuda_plan_forward));
    is_fft_planned = false;
  }

//  if (is_cublas_loaded)
//  {
//    checkCudaErrors(hipblasDestroy(cublasHandle));
//    is_cublas_loaded = false;
//  }

  if (is_allocated_mask_CSOS)
  {
    mask_CSOS->Deallocate();
  }

  if (is_allocated_image_buffer)
  {
    image_buffer->Deallocate();
  }

  if (is_allocated_sum_buffer) checkCudaErrors(hipFree(this->sum_buffer)); is_allocated_sum_buffer = false;


}

void GpuImage::ConvertToHalfPrecision(bool deallocate_single_precision)
{

	// FIXME when adding real space complex images
	MyAssertTrue( is_in_memory_gpu, "Image is in not on the GPU!");

	BufferInit(b_16f);

	pre_checkErrorsAndTimingWithSynchronization(hipStreamPerThread);
	ReturnLaunchParamters(dims, false);
	ConvertToHalfPrecisionKernel<< <gridDims, threadsPerBlock,0, hipStreamPerThread>> > (complex_values_gpu, complex_values_16f, this->dims, this->physical_upper_bound_complex);
	checkErrorsAndTimingWithSynchronization(hipStreamPerThread);

	if (deallocate_single_precision)
	{
		checkCudaErrors(hipFree(real_values_gpu));
		is_in_memory_gpu = false;
	}
}

__global__ void ConvertToHalfPrecisionKernel(hipfftComplex* complex_32f_values, __half2* complex_16f_values, int4 dims, int3 physical_upper_bound_complex)
{

	int3 coords = make_int3(blockIdx.x*blockDim.x + threadIdx.x,
						  blockIdx.y*blockDim.y + threadIdx.y,
						  blockIdx.z);

	if (coords.x < dims.w / 2 && coords.y < dims.y && coords.z < dims.z)
	{

		int address = d_ReturnFourier1DAddressFromPhysicalCoord(coords, physical_upper_bound_complex);

		complex_16f_values[address] = __float22half2_rn(complex_32f_values[address]);
	}


}
void GpuImage::Allocate(int wanted_x_size, int wanted_y_size, int wanted_z_size, bool should_be_in_real_space)
{

	MyAssertTrue(wanted_x_size > 0 && wanted_y_size > 0 && wanted_z_size > 0,"Bad dimensions: %i %i %i\n",wanted_x_size,wanted_y_size,wanted_z_size);

	// check to see if we need to do anything?

	if (is_in_memory_gpu == true)
	{
		is_in_real_space = should_be_in_real_space;
		if (wanted_x_size == dims.x && wanted_y_size == dims.y && wanted_z_size == dims.z)
		{
			// everything is already done..
			is_in_real_space = should_be_in_real_space;
	//			wxPrintf("returning\n");
			return;
		}
		else
		{
		  Deallocate();
		}
	}

	SetupInitialValues();
	this->is_in_real_space = should_be_in_real_space;
	dims.x = wanted_x_size; dims.y = wanted_y_size; dims.z = wanted_z_size;

	// if we got here we need to do allocation..

	// first_x_dimension
	if (IsEven(wanted_x_size) == true) real_memory_allocated =  wanted_x_size / 2 + 1;
	else real_memory_allocated = (wanted_x_size - 1) / 2 + 1;

	real_memory_allocated *= wanted_y_size * wanted_z_size; // other dimensions
	real_memory_allocated *= 2; // room for complex

	// TODO consider option to add host mem here. For now, just do gpu mem.
	//////	real_values = (float *) fftwf_malloc(sizeof(float) * real_memory_allocated);
	//////	complex_values = (std::complex<float>*) real_values;  // Set the complex_values to point at the newly allocated real values;
//	wxPrintf("\n\n\tAllocating mem\t\n\n");
	checkCudaErrors(hipMalloc(&real_values_gpu, real_memory_allocated*sizeof(hipfftReal)));
	complex_values_gpu = (hipfftComplex *)real_values_gpu;
	is_in_memory_gpu = true;

	// Update addresses etc..
	UpdateLoopingAndAddressing(wanted_x_size, wanted_y_size, wanted_z_size);

	if (IsEven(wanted_x_size) == true) padding_jump_value = 2;
	else padding_jump_value = 1;

	// record the full length ( pitch / 4 )
	dims.w = dims.x + padding_jump_value;
	pitch = dims.w * sizeof(float);

	number_of_real_space_pixels = int(dims.x) * int(dims.y) * int(dims.z);
	ft_normalization_factor = 1.0 / sqrtf(float(number_of_real_space_pixels));


	// Set other gpu vals

	is_host_memory_pinned = false;
	is_meta_data_initialized = true;

}

void GpuImage::UpdateBoolsToDefault()
{

	is_meta_data_initialized = false;

	is_in_memory = false;
	is_in_real_space = true;
	object_is_centred_in_box = true;
	image_memory_should_not_be_deallocated = false;

	is_in_memory_gpu = false;
	is_host_memory_pinned = false;

	// libraries
	is_fft_planned = false;
//	is_cublas_loaded = false;
	is_npp_loaded = false;

	// Buffers
	is_allocated_image_buffer = false;
	is_allocated_mask_CSOS = false;

	is_allocated_sum_buffer = false;
	is_allocated_min_buffer = false;
	is_allocated_minIDX_buffer = false;
	is_allocated_max_buffer = false;
	is_allocated_maxIDX_buffer = false;
	is_allocated_minmax_buffer = false;
	is_allocated_minmaxIDX_buffer = false;
	is_allocated_mean_buffer = false;
	is_allocated_meanstddev_buffer = false;
	is_allocated_countinrange_buffer = false;
	is_allocated_l2norm_buffer = false;
	is_allocated_dotproduct_buffer = false;
	is_allocated_16f_buffer = false;

	// Callbacks
	is_set_convertInputf16Tof32 = false;
	is_set_scaleFFTAndStore = false;
	is_set_complexConjMulLoad = false;
	is_set_realLoadAndClipInto = false;

}

//!>  \brief  Update all properties related to looping & addressing in real & Fourier space, given the current logical dimensions.

void GpuImage::UpdateLoopingAndAddressing(int wanted_x_size, int wanted_y_size, int wanted_z_size)
{


	dims.x = wanted_x_size;
	dims.y = wanted_y_size;
	dims.z = wanted_z_size;

	physical_address_of_box_center.x = wanted_x_size / 2;
	physical_address_of_box_center.y= wanted_y_size / 2;
	physical_address_of_box_center.z= wanted_z_size / 2;

	physical_upper_bound_complex.x= wanted_x_size / 2;
	physical_upper_bound_complex.y= wanted_y_size - 1;
	physical_upper_bound_complex.z= wanted_z_size - 1;


	//physical_index_of_first_negative_frequency.x= wanted_x_size / 2 + 1;
	if (IsEven(wanted_y_size) == true)
	{
		physical_index_of_first_negative_frequency.y= wanted_y_size / 2;
	}
	else
	{
		physical_index_of_first_negative_frequency.y= wanted_y_size / 2 + 1;
	}

	if (IsEven(wanted_z_size) == true)
	{
		physical_index_of_first_negative_frequency.z= wanted_z_size / 2;
	}
	else
	{
		physical_index_of_first_negative_frequency.z= wanted_z_size / 2 + 1;
	}


    // Update the Fourier voxel size

	fourier_voxel_size.x= 1.0 / double(wanted_x_size);
	fourier_voxel_size.y= 1.0 / double(wanted_y_size);
	fourier_voxel_size.z= 1.0 / double(wanted_z_size);

	// Logical bounds
	if (IsEven(wanted_x_size) == true)
	{
		logical_lower_bound_complex.x= -wanted_x_size / 2;
		logical_upper_bound_complex.x=  wanted_x_size / 2;
	    logical_lower_bound_real.x   = -wanted_x_size / 2;
	    logical_upper_bound_real.x   =  wanted_x_size / 2 - 1;
	}
	else
	{
		logical_lower_bound_complex.x= -(wanted_x_size-1) / 2;
		logical_upper_bound_complex.x=  (wanted_x_size-1) / 2;
		logical_lower_bound_real.x   = -(wanted_x_size-1) / 2;
		logical_upper_bound_real.x    =  (wanted_x_size-1) / 2;
	}


	if (IsEven(wanted_y_size) == true)
	{
	    logical_lower_bound_complex.y= -wanted_y_size / 2;
	    logical_upper_bound_complex.y=  wanted_y_size / 2 - 1;
	    logical_lower_bound_real.y   = -wanted_y_size / 2;
	    logical_upper_bound_real.y   =  wanted_y_size / 2 - 1;
	}
	else
	{
	    logical_lower_bound_complex.y= -(wanted_y_size-1) / 2;
	    logical_upper_bound_complex.y=  (wanted_y_size-1) / 2;
	    logical_lower_bound_real.y   = -(wanted_y_size-1) / 2;
	    logical_upper_bound_real.y    =  (wanted_y_size-1) / 2;
	}

	if (IsEven(wanted_z_size) == true)
	{
		logical_lower_bound_complex.z= -wanted_z_size / 2;
		logical_upper_bound_complex.z=  wanted_z_size / 2 - 1;
		logical_lower_bound_real.z   = -wanted_z_size / 2;
		logical_upper_bound_real.z   =  wanted_z_size / 2 - 1;

	}
	else
	{
		logical_lower_bound_complex.z= -(wanted_z_size - 1) / 2;
		logical_upper_bound_complex.z=  (wanted_z_size - 1) / 2;
		logical_lower_bound_real.z   = -(wanted_z_size - 1) / 2;
		logical_upper_bound_real.z    =  (wanted_z_size - 1) / 2;
	}
}



